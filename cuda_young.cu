

#include <hip/hip_runtime.h>
#include <stdio.h>

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

__global__ void transposeCoalesced(double *odata, const double *idata, int rows,int cols)
{
  	__shared__ double tile[TILE_DIM][TILE_DIM+1];

  	int x = blockIdx.x * TILE_DIM + threadIdx.x;
  	int y = blockIdx.y * TILE_DIM + threadIdx.y;

	//  if (x >= cols||y >= rows){
	//      return;
	//  }

  	int maxJ = TILE_DIM;
  	int maxJ2 = TILE_DIM;
  	int otherMaxJ = rows - y;
  	if (maxJ > otherMaxJ)
    	maxJ = otherMaxJ;


  	if ( x < cols ){
    		for (int j = 0; j < maxJ; j += BLOCK_ROWS)
     		tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*cols + x];
  	}
  	__syncthreads();

  	x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  	y = blockIdx.x * TILE_DIM + threadIdx.y;

  	int otherMaxJ2 = cols - y;
  	if (maxJ2 > otherMaxJ2){
      	maxJ2 = otherMaxJ2;
  	}
  	if ( x < rows){
   		for (int j = 0; j < maxJ2; j += BLOCK_ROWS)
      	 	odata[(y+j)*rows + x] = tile[threadIdx.x][threadIdx.y + j];
  	}

}



__global__ void cuconvolve_youngCausal(double * in, double * out, int rows, int columns, double B, double *bf) 
{    


	unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;

    	if(idx<columns)
	{
    		/* Compute first 3 output elements */  
    		out[idx] = B*in[idx];

    		out[idx+columns] = B*in[idx+columns] + bf[2]*out[idx];

    		out[idx+2*columns] = B*in[idx+2*columns] + (bf[1]*out[idx]+bf[2]*out[idx+columns]);
    
    		/* Recursive computation of output in forward direction using filter parameters bf and B */
    		for(int i=3; i<rows; i++) 
   		{
        		out[idx+i*columns] = B*in[idx+i*columns];

        		for(int j=0; j<3; j++) 
	  		{
            		out[idx+i*columns] += bf[j]*out[idx + (i-(3-j))*columns];
        		}
    		}

	}  
}

__global__ void cuconvolve_youngAnticausal(double * in, double * out, int rows, int columns, double B, double *bb) 
{
	unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
    
    	int total = columns*(rows-1);

    	if(idx<columns)
	{
    		/* Compute last 3 output elements */
    		out[total + idx] = B*in[total + idx];

    		out[total + idx - columns] = B*in[total + idx - columns] + bb[0]*out[total + idx];

    		out[total + idx - 2*columns] = B*in[total + idx - 2*columns] + (bb[0]*out[total + idx - columns]+bb[1]*out[total + idx]);
    
    		/* Recursive computation of output in backward direction using filter parameters bb and B */
    		for (int i=3; i<rows-1; i++) 
    		{
        		out[total + idx - i*columns] = B*in[total + idx - i*columns];
        		for (int j=0; j<3; j++) 
	  		{
            		out[total + idx - i*columns] += bb[j]*out[total + idx - (i-(j+1))*columns];
        		}
    		}
   	}       
}

extern "C"
void cudaYoung(double * in, double * out, int rows, int columns, double *bf, double *bb, double B) 
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

    /** \brief Array to store output of Causal filter convolution */

	double *d_input, *d_output, *d_bf, *d_bb;
	hipMalloc((void**) &d_input, rows*columns*sizeof(double)); 
	hipMalloc((void**) &d_output, rows*columns*sizeof(double)); 
	
	hipMalloc((void**) &d_bf, rows*columns*sizeof(double));
	hipMalloc((void**) &d_bb, rows*columns*sizeof(double));

	hipMemcpy(d_input, in, rows*columns*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_bf, bf, 3*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_bb, bb, 3*sizeof(double), hipMemcpyHostToDevice);


	dim3 dimGrid1((columns+TILE_DIM-1)/TILE_DIM,(rows+TILE_DIM-1)/TILE_DIM, 1);
	dim3 dimGrid2((rows+TILE_DIM-1)/TILE_DIM,(columns+TILE_DIM-1)/TILE_DIM, 1);
	dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

	// -------- Convolve Rows----------

	transposeCoalesced<<< dimGrid1, dimBlock>>>(d_output, d_input, rows, columns);

    	cuconvolve_youngCausal<<<rows/256 + 1 , 256>>>(d_output, d_input, columns, rows, B, d_bf);

    	cuconvolve_youngAnticausal<<<rows/256 + 1, 256>>>(d_input, d_output, columns, rows, B, d_bb);

	// -------- Convolve Columns ----------

	transposeCoalesced<<< dimGrid2, dimBlock>>>(d_input, d_output, columns, rows);

    	cuconvolve_youngCausal<<<columns/256 + 1, 256>>>(d_input, d_output, rows, columns, B, d_bf);

    	cuconvolve_youngAnticausal<<<columns/256 + 1, 256>>>(d_output, d_input, rows, columns, B, d_bb);

	hipMemcpy(in, d_input, rows*columns*sizeof(double), hipMemcpyDeviceToHost);

	hipEventRecord(stop);

  	hipEventSynchronize(stop);
  	float milliseconds = 0;
  	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Execution time elapsed: %f ms\n", milliseconds);

	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_bf);
	hipFree(d_bb);
}



